#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
__global__ cudas(char* pA, char pB, int n){

	idx= threadIdx.x;
	pB[n-idx-1]=pA[idx];
}
int main(){
	char a[50];
	char b[50];
	char *pA;
	char *pB;
	printf("enter the string\n");
	scanf("%s",a);
	int n=strlen(a);
	hipMalloc((void**)&pA,n*sizeof(char));
	hipMalloc((void**)&pB,n*sizeof(char));
	hipMemcpy(pA,a,n*sizeof(char),hipMemcpyHostToDevice);
	dim3_t dim;
	dim=n;
	cudas<<<1,dim>>>(pa,pb,n);
	hipMemcpy(pb,b,n*sizeof(char,hipMemcpyDeviceToHost);
	printf("after revers, %s",b);
	hipFree(pA);
	hipFree(pB);
	return 0;

}